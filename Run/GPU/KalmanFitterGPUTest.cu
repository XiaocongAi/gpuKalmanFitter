#include "hip/hip_runtime.h"
#include "EventData/PixelSourceLink.hpp"
#include "EventData/TrackParameters.hpp"
#include "Fitter/GainMatrixUpdater.hpp"
#include "Fitter/KalmanFitter.hpp"
#include "Material/HomogeneousSurfaceMaterial.hpp"
#include "Propagator/EigenStepper.hpp"
#include "Propagator/Propagator.hpp"
#include "Utilities/CudaHelper.hpp"
#include "Utilities/ParameterDefinitions.hpp"
#include "Utilities/Profiling.hpp"
#include "Utilities/Units.hpp"

#include "ActsExamples/Generator.hpp"
#include "ActsExamples/MultiplicityGenerators.hpp"
#include "ActsExamples/ParametricParticleGenerator.hpp"
#include "ActsExamples/RandomNumbers.hpp"
#include "ActsExamples/VertexGenerators.hpp"

#include "Test/Helper.hpp"
#include "Test/Logger.hpp"

#include "Processor.hpp"

#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

// This executable is used to run the KalmanFitter fit test on GPU with
// parallelism on the track-level. It contains mainly two parts: 1) Explicit
// calling of the propagation to create measurements on tracks ( a 'simulated'
// track could contain 10~100 measurements) 2) Running the Kalmanfitter using
// the created measurements in 1) as one of the inputs In princinple, both 1)
// and 2) could on offloaded to GPU. Right now, only 2) is put into a kernel

static void show_usage(std::string name) {
  std::cerr << "Usage: <option(s)> VALUES"
            << "Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-t,--tracks \tSpecify the number of tracks\n"
            << "\t-r,--streams \tSpecify number of streams\n"
            // << "\t-p,--pt \tSpecify the pt of particle\n"
            << "\t-o,--output \tIndicator for writing propagation results\n"
            << "\t-d,--device \tSpecify the device: 'gpu' or 'cpu'\n"
            << "\t-g,--grid-size \tSpecify GPU grid size: 'x*y'\n"
            << "\t-b,--block-size \tSpecify GPU block size: 'x*y*z'\n"
            << "\t-s,--shared-memory \tIndicator for using shared memory for "
               "one track or not\n"
            << std::endl;
}

using Stepper = Acts::EigenStepper<Test::ConstantBField>;
using PropagatorType = Acts::Propagator<Stepper>;
using PropResultType = Acts::PropagatorResult;
using PropOptionsType = Acts::PropagatorOptions<Simulator, Test::VoidAborter>;
using PropState = PropagatorType::State<PropOptionsType>;
using KalmanFitterType =
    Acts::KalmanFitter<PropagatorType, Acts::GainMatrixUpdater>;
using KalmanFitterResultType =
    Acts::KalmanFitterResult<Acts::PixelSourceLink, Acts::BoundParameters>;
using TSType = typename KalmanFitterResultType::TrackStateType;

// Device code
__global__ void __launch_bounds__(256, 2) fitKernelThreadPerTrack(
    KalmanFitterType *kFitter, Acts::PixelSourceLink *sourcelinks,
    Acts::CurvilinearParameters *tpars,
    Acts::KalmanFitterOptions<Acts::VoidOutlierFinder> kfOptions,
    TSType *fittedTracks, bool *fitStatus, const Acts::Surface *surfacePtrs,
    int nSurfaces, int nTracks, int offset) {
  // In case of 1D grid and 1D block, the threadId = blockDim.x*blockIdx.x +
  // threadIdx.x + offset
  // @note This might have problem if the number of threads is smaller than the
  // number of tracks!!!
  int threadId =
      blockDim.x * blockDim.y * (gridDim.x * blockIdx.y + blockIdx.x) +
      blockDim.x * threadIdx.y + threadIdx.x + offset;

  // Different threads handles different track
  if (threadId < (nTracks + offset)) {
    // Use the CudaKernelContainer for the source links and fitted tracks
    KalmanFitterResultType kfResult;
    kfResult.fittedStates = CudaKernelContainer<TSType>(
        fittedTracks + threadId * nSurfaces, nSurfaces);
    fitStatus[threadId] = kFitter->fit(
        Acts::CudaKernelContainer<PixelSourceLink>(
            sourcelinks + threadId * nSurfaces, nSurfaces),
        tpars[threadId], kfOptions, kfResult, surfacePtrs, nSurfaces);
  }
}

__global__ void __launch_bounds__(256, 2) fitKernelBlockPerTrack(
    KalmanFitterType *kFitter, Acts::PixelSourceLink *sourcelinks,
    Acts::CurvilinearParameters *tpars,
    Acts::KalmanFitterOptions<Acts::VoidOutlierFinder> kfOptions,
    TSType *fittedTracks, bool *fitStatus, const Acts::Surface *surfacePtrs,
    int nSurfaces, int nTracks, int offset) {
  // @note This will have problem if the number of blocks is smaller than the
  // number of tracks!!!
  int blockId = gridDim.x * blockIdx.y + blockIdx.x + offset;

  // All threads in this block handles the same track
  if (blockId < (nTracks + offset)) {
    // Use the CudaKernelContainer for the source links and fitted tracks
    KalmanFitterResultType kfResult;
    kfResult.fittedStates = CudaKernelContainer<TSType>(
        fittedTracks + blockId * nSurfaces, nSurfaces);
    fitStatus[blockId] = kFitter->fitOnDevice(
        Acts::CudaKernelContainer<PixelSourceLink>(
            sourcelinks + blockId * nSurfaces, nSurfaces),
        tpars[blockId], kfOptions, kfResult, surfacePtrs, nSurfaces);
  }
}

int main(int argc, char *argv[]) {
  unsigned int nTracks = 10240;
  unsigned int nStreams = 1;
  bool output = false;
  bool useSharedMemory = false;
  std::string device = "cpu";
  std::string bFieldFileName;
  // double p = 1 * Acts::units::_GeV;
  dim3 grid(20000), block(8, 8);
  // This should always be included
  for (int i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if ((arg == "-h") or (arg == "--help")) {
      show_usage(argv[0]);
      return 0;
    } else if (i + 1 < argc) {
      if ((arg == "-t") or (arg == "--tracks")) {
        nTracks = atoi(argv[++i]);
      } else if ((arg == "-r") or (arg == "--streams")) {
        nStreams = atoi(argv[++i]);
        //} else if ((arg == "-p") or (arg == "--pt")) {
        //  p = atof(argv[++i]) * Acts::units::_GeV;
      } else if ((arg == "-o") or (arg == "--output")) {
        output = (atoi(argv[++i]) == 1);
      } else if ((arg == "-d") or (arg == "--device")) {
        device = argv[++i];
      } else if ((arg == "-g") or (arg == "--grid-size")) {
        grid = stringToDim3(argv[++i]);
      } else if ((arg == "-b") or (arg == "--block-size")) {
        block = stringToDim3(argv[++i]);
      } else if ((arg == "-s") or (arg == "--shared-memory")) {
        useSharedMemory = (atoi(argv[++i]) == 1);
      } else {
        std::cerr << "Unknown argument." << std::endl;
        return 1;
      }
    }
  }

  if (grid.z != 1 or block.z != 1) {
    std::cout << "3D grid or block is not supported at the moment! Good luck!"
              << std::endl;
    return 1;
  }
  std::cout << grid.x << " " << grid.y << " " << block.x << " " << block.y
            << std::endl;

  int devId = 0;

  hipDeviceProp_t prop;
  GPUERRCHK(hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  GPUERRCHK(hipSetDevice(devId));
  int driverVersion, rtVersion;
  GPUERRCHK(hipDriverGetVersion(&driverVersion));
  printf("cuda driver version: %i\n", driverVersion);
  GPUERRCHK(hipRuntimeGetVersion(&rtVersion));
  printf("cuda rt version: %i\n", rtVersion);

  int tracksPerBlock = block.x * block.y;

  // Use 8*8 block if using one block for one track
  // @todo Extend to run multiple (block.z) tracks in one block
  if (useSharedMemory) {
    std::cout << "Shared memory used. Block size is set to 8*8!" << std::endl;
    block = dim3(8, 8);
    tracksPerBlock = 1;
  }

  // The last stream could could less tracks
  const unsigned int tracksPerStream = (nTracks + nStreams - 1) / nStreams;
  const unsigned int tracksLastStream =
      tracksPerStream - (tracksPerStream * nStreams - nTracks);
  std::cout << "tracksPerStream : tracksLastStream = " << tracksPerStream
            << " : " << tracksLastStream << std::endl;

  // @note shall we use this for the grid size?
  const unsigned int blocksPerGrid =
      (tracksPerStream + tracksPerBlock - 1) / tracksPerBlock;
  if (grid.x * grid.y < blocksPerGrid) {
    std::cout << "Grid size too small. It should be at least " << blocksPerGrid
              << std::endl;
    return 1;
  }

  // The shared memory size
  int sharedMemoryPerTrack = sizeof(PathLimitReached) + sizeof(PropState) +
                             sizeof(bool) * 2 + sizeof(PropagatorResult);
  std::cout << "shared memory is " << sharedMemoryPerTrack << std::endl;

  // The number of test surfaces
  const unsigned int nSurfaces = 10;
  const unsigned int surfaceBytes = sizeof(PlaneSurfaceType) * nSurfaces;
  const unsigned int sourcelinksBytes =
      sizeof(PixelSourceLink) * nSurfaces * nTracks;
  const unsigned int parsBytes = sizeof(CurvilinearParameters) * nTracks;
  const unsigned int tsBytes = sizeof(TSType) * nSurfaces * nTracks;
  const unsigned int statusBytes = sizeof(bool) * nTracks;
  std::cout << "surface Bytes = " << surfaceBytes << std::endl;
  std::cout << "source links Bytes = " << sourcelinksBytes << std::endl;
  std::cout << "startPars Bytes = " << parsBytes << std::endl;
  std::cout << "TSs Bytes = " << tsBytes << std::endl;

  const unsigned int perSourcelinksBytes =
      sizeof(PixelSourceLink) * nSurfaces * tracksPerStream;
  const unsigned int lastSourcelinksBytes =
      sizeof(PixelSourceLink) * nSurfaces * tracksLastStream;
  const unsigned int perParsBytes =
      sizeof(CurvilinearParameters) * tracksPerStream;
  const unsigned int lastParsBytes =
      sizeof(CurvilinearParameters) * tracksLastStream;
  const unsigned int perTSsBytes = sizeof(TSType) * nSurfaces * tracksPerStream;
  const unsigned int lastTSsBytes =
      sizeof(TSType) * nSurfaces * tracksLastStream;
  const unsigned int perStatusBytes = sizeof(bool) * tracksPerStream;
  const unsigned int lastStatusBytes = sizeof(bool) * tracksLastStream;

  // Create a test context
  Acts::GeometryContext gctx(0);
  Acts::MagneticFieldContext mctx(0);

  // Create a random number service
  ActsExamples::RandomNumbers::Config config;
  auto randomNumbers = std::make_shared<ActsExamples::RandomNumbers>(config);
  auto rng = randomNumbers->spawnGenerator(0);

  // Create the geometry
  // Set translation vectors
  std::vector<Acts::Vector3D> translations;
  for (unsigned int isur = 0; isur < nSurfaces; isur++) {
    translations.push_back({(isur * 30. + 19) * Acts::units::_mm, 0., 0.});
  }
  // The silicon material
  Acts::MaterialSlab matProp(Test::makeSilicon(), 0.5 * Acts::units::_mm);
  if (matProp) {
    std::cout << "matProp has material" << std::endl;
  }
  Acts::HomogeneousSurfaceMaterial surfaceMaterial(matProp);
  if (surfaceMaterial.materialSlab()) {
    std::cout << "surfaceMaterial has material" << std::endl;
  }
  // Create plane surfaces without boundaries
  PlaneSurfaceType *surfaces;
  // Unified memory allocation for geometry
  GPUERRCHK(hipMallocManaged(&surfaces, sizeof(PlaneSurfaceType) * nSurfaces));
  std::cout << "Allocating the memory for the surfaces" << std::endl;
  for (unsigned int isur = 0; isur < nSurfaces; isur++) {
    surfaces[isur] = PlaneSurfaceType(translations[isur],
                                      Acts::Vector3D(1, 0, 0), surfaceMaterial);
    if (surfaces[isur].surfaceMaterial().materialSlab()) {
      std::cout << "has material " << std::endl;
    }
  }
  const Acts::Surface *surfacePtrs = surfaces;
  std::cout << "Creating " << nSurfaces << " boundless plane surfaces"
            << std::endl;

  // Prepare to run the particles generation
  ActsExamples::GaussianVertexGenerator vertexGen;
  vertexGen.stddev[Acts::eFreePos0] = 1.0 * Acts::units::_mm;
  vertexGen.stddev[Acts::eFreePos1] = 1.0 * Acts::units::_mm;
  vertexGen.stddev[Acts::eFreePos2] = 5.0 * Acts::units::_mm;
  vertexGen.stddev[Acts::eFreeTime] = 1.0 * Acts::units::_ns;
  ActsExamples::ParametricParticleGenerator::Config pgCfg;
  // @note We are generating 20% more particles to make sure we could get enough
  // valid particles
  size_t nGeneratedParticles = nTracks * 1.2;
  ActsExamples::Generator generator = ActsExamples::Generator{
      ActsExamples::FixedMultiplicityGenerator{nGeneratedParticles},
      std::move(vertexGen), ActsExamples::ParametricParticleGenerator(pgCfg)};
  // Run the generation to generate particles
  std::vector<ActsFatras::Particle> generatedParticles;
  runParticleGeneration(rng, generator, generatedParticles);

  // Prepare to run the simulation
  Stepper stepper;
  PropagatorType propagator(stepper);
  auto start_propagate = std::chrono::high_resolution_clock::now();
  std::cout << "start to run propagation" << std::endl;
  // Run the simulation to generate sim hits
  // @note We will pick up the valid particles
  std::vector<Simulator::result_type> simResult(nTracks);
  std::vector<ActsFatras::Particle> validParticles(nTracks);
  runSimulation(gctx, mctx, rng, propagator, generatedParticles, validParticles,
                simResult, surfacePtrs, nSurfaces);
  auto end_propagate = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_seconds =
      end_propagate - start_propagate;
  std::cout << "Time (ms) to run propagation tests: "
            << elapsed_seconds.count() * 1000 << std::endl;
  if (output) {
    std::cout << "writing propagation results" << std::endl;
    Test::writeSimHits(simResult);
  }

  // The hit smearing resolution
  std::array<double, 2> hitResolution = {30. * Acts::units::_mm,
                                         30. * Acts::units::_mm};
  // Pinned memory for source links
  Acts::PixelSourceLink *sourcelinks;
  GPUERRCHK(hipHostMalloc((void **)&sourcelinks, sourcelinksBytes));
  // Run hit smearing to create source links
  // @note pass the concreate PlaneSurfaceType pointer here
  runHitSmearing(gctx, rng, simResult, hitResolution, sourcelinks, surfaces,
                 nSurfaces);

  // The particle smearing resolution
  ParticleSmearingParameters seedResolution;
  // Run truth seed smearing to create starting parameters
  auto startParsCollection =
      runParticleSmearing(rng, gctx, validParticles, seedResolution, nTracks);
  // Pinned memory for starting track parameters to be transferred to GPU
  CurvilinearParameters *startPars;
  GPUERRCHK(hipHostMalloc((void **)&startPars, parsBytes));
  // Copy to the pinned memory
  memcpy(startPars, startParsCollection.data(), parsBytes);

  // Prepare to perform fit to the created tracks
  KalmanFitterType kFitter(propagator);
  KalmanFitterOptions<VoidOutlierFinder> kfOptions(gctx, mctx);
  // KalmanFitterOptions<VoidOutlierFinder> kfOptions(
  //    gctx, mctx, Acts::VoidOutlierFinder(), nullptr, multipleScattering,
  //    energyLoss);
  // Pinned memory for KF fitted tracks
  TSType *fittedTracks;
  GPUERRCHK(hipHostMalloc((void **)&fittedTracks, tsBytes));
  // Pinned memory for KF fit status
  bool *fitStatus;
  GPUERRCHK(hipHostMalloc((void **)&fitStatus, statusBytes));

  float ms; // elapsed time in milliseconds

  // Create events and streams
  hipEvent_t startEvent, stopEvent;
  hipStream_t stream[nStreams];
  GPUERRCHK(hipEventCreate(&startEvent));
  GPUERRCHK(hipEventCreate(&stopEvent));
  for (int i = 0; i < nStreams; ++i) {
    GPUERRCHK(hipStreamCreate(&stream[i]));
  }

  // Running directly on host or offloading to GPU
  bool useGPU = (device == "gpu");
  if (useGPU) {
    GPUERRCHK(hipEventRecord(startEvent, 0));

    // Allocate memory on device
    PixelSourceLink *d_sourcelinks;
    CurvilinearParameters *d_pars;
    KalmanFitterType *d_kFitter;
    TSType *d_fittedTracks;
    bool *d_fitStatus;
    GPUERRCHK(hipMalloc(&d_sourcelinks, sourcelinksBytes));
    GPUERRCHK(hipMalloc(&d_pars, parsBytes));
    GPUERRCHK(hipMalloc(&d_fittedTracks, tsBytes));
    GPUERRCHK(hipMalloc(&d_fitStatus, statusBytes));
    GPUERRCHK(hipMalloc(&d_kFitter, sizeof(KalmanFitterType)));

    // Copy the KalmanFitter from host to device (shared between all tracks)
    GPUERRCHK(hipMemcpy(d_kFitter, &kFitter, sizeof(KalmanFitterType),
                         hipMemcpyHostToDevice));

    // Run on device
    // for (int _ : {1, 2, 3, 4, 5}) {
    for (unsigned int i = 0; i < nStreams; ++i) {
      unsigned int offset = i * tracksPerStream;
      // The number of tracks handled in this stream
      unsigned int streamTracks = tracksPerStream;
      unsigned int sBytes = perSourcelinksBytes;
      unsigned int pBytes = perParsBytes;
      unsigned int tBytes = perTSsBytes;
      unsigned int stBytes = perStatusBytes;
      if (i == (nStreams - 1)) {
        streamTracks = tracksLastStream;
        sBytes = lastSourcelinksBytes;
        pBytes = lastParsBytes;
        tBytes = lastTSsBytes;
        stBytes = lastStatusBytes;
      }

      if (i == 0) {
        // @note: prefetch the surface or not
        hipMemPrefetchAsync(surfaces, surfaceBytes, devId, stream[i]);
      }

      // Copy the sourcelinsk, starting parameters and fitted tracks from host
      // to device
      GPUERRCHK(hipMemcpyAsync(&d_sourcelinks[offset * nSurfaces],
                                &sourcelinks[offset * nSurfaces], sBytes,
                                hipMemcpyHostToDevice, stream[i]));
      GPUERRCHK(hipMemcpyAsync(&d_pars[offset], &startPars[offset], pBytes,
                                hipMemcpyHostToDevice, stream[i]));
      GPUERRCHK(hipMemcpyAsync(&d_fittedTracks[offset], &fittedTracks[offset],
                                tBytes, hipMemcpyHostToDevice, stream[i]));
      GPUERRCHK(hipMemcpyAsync(&d_fitStatus[offset], &fitStatus[offset],
                                stBytes, hipMemcpyHostToDevice, stream[i]));

      // Use shared memory for one track if requested
      if (useSharedMemory) {
        fitKernelBlockPerTrack<<<grid, block, 0, stream[i]>>>(
            d_kFitter, d_sourcelinks, d_pars, kfOptions, d_fittedTracks,
            d_fitStatus, surfacePtrs, nSurfaces, streamTracks, offset);
      } else {
        fitKernelThreadPerTrack<<<grid, block, 0, stream[i]>>>(
            d_kFitter, d_sourcelinks, d_pars, kfOptions, d_fittedTracks,
            d_fitStatus, surfacePtrs, nSurfaces, streamTracks, offset);
      }
      GPUERRCHK(hipEventRecord(stopEvent, stream[i]));
      GPUERRCHK(hipEventSynchronize(stopEvent));
      // copy the fitted tracks to host
      GPUERRCHK(hipMemcpyAsync(&fittedTracks[offset], &d_fittedTracks[offset],
                                tBytes, hipMemcpyDeviceToHost, stream[i]));
      GPUERRCHK(hipMemcpyAsync(&fitStatus[offset], &d_fitStatus[offset],
                                stBytes, hipMemcpyDeviceToHost, stream[i]));
    }

    GPUERRCHK(hipPeekAtLastError());
    GPUERRCHK(hipDeviceSynchronize());

    // Free the memory on device
    GPUERRCHK(hipFree(d_sourcelinks));
    GPUERRCHK(hipFree(d_pars));
    GPUERRCHK(hipFree(d_fittedTracks));
    GPUERRCHK(hipFree(d_fitStatus));
    GPUERRCHK(hipFree(d_kFitter));

    GPUERRCHK(hipEventRecord(stopEvent, 0));
    GPUERRCHK(hipEventSynchronize(stopEvent));
    GPUERRCHK(hipEventElapsedTime(&ms, startEvent, stopEvent));
    printf("Time (ms) for KF memory transfer and execution: %f\n", ms);

    // Log the execution time in seconds (not including the managed memory
    // allocation time for the surfaces)
    Test::Logger::logTime(Test::Logger::buildFilename(
                              "timing_gpu", "nTracks", std::to_string(nTracks),
                              "gridSize", dim3ToString(grid), "blockSize",
                              dim3ToString(block)),
                          ms / 1000);

  } else {
    /// Run on host
    auto start_fit = std::chrono::high_resolution_clock::now();
#pragma omp parallel for num_threads(250)
    for (int it = 0; it < nTracks; it++) {
      // The fit result wrapper
      KalmanFitterResultType kfResult;
      kfResult.fittedStates = Acts::CudaKernelContainer<TSType>(
          &fittedTracks[it * nSurfaces], nSurfaces);
      // The input source links wrapper
      auto sourcelinkTrack = Acts::CudaKernelContainer<PixelSourceLink>(
          sourcelinks + it * nSurfaces, nSurfaces);
      // Run the fit. The fittedTracks will be changed here
      auto status = kFitter.fit(sourcelinkTrack, startParsCollection[it],
                                kfOptions, kfResult, surfacePtrs, nSurfaces);
      if (not status) {
        std::cout << "fit failure for track " << it << std::endl;
      }
      fitStatus[it] = status;
    }
    auto end_fit = std::chrono::high_resolution_clock::now();
    elapsed_seconds = end_fit - start_fit;
    std::cout << "Time (ms) to run KalmanFitter for " << nTracks << " : "
              << elapsed_seconds.count() * 1000 << std::endl;
  }
  int threads = omp_get_num_threads();

  if (output) {
    std::cout << "writing KF results" << std::endl;
    std::string fileName;
    if (useGPU) {
      fileName = "fitted_tracks_gpu_nTracks_";
    } else {
      fileName = "fitted_tracks_semi_cpu_nTracks_";
    }
    fileName.append(std::to_string(nTracks)).append(".obj");
    Test::writeTracks(fittedTracks, fitStatus, nTracks, nSurfaces, fileName);
  }

  std::cout << "------------------------  ending  -----------------------"
            << std::endl;

  // Free the managed/pinned memory
  GPUERRCHK(hipFree(surfaces));
  GPUERRCHK(hipHostFree(sourcelinks));
  GPUERRCHK(hipHostFree(startPars));
  GPUERRCHK(hipHostFree(fittedTracks));
  GPUERRCHK(hipHostFree(fitStatus));

  return 0;
}

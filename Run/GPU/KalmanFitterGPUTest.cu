#include "hip/hip_runtime.h"
#include "FitData.hpp"
#include "Processor.hpp"
#include "Writer.hpp"

#include "Geometry/GeometryID.hpp"
#include "Material/HomogeneousSurfaceMaterial.hpp"
#include "Utilities/CudaHelper.hpp"
#include "Utilities/Profiling.hpp"

#include "ActsExamples/MultiplicityGenerators.hpp"
#include "ActsExamples/ParametricParticleGenerator.hpp"
#include "ActsExamples/VertexGenerators.hpp"

#include "Test/Helper.hpp"
#include "Test/Logger.hpp"

#include "DataSizeCalculator.cu"
#include "Kernels.cu"

#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

// This executable is used to run the KalmanFitter fit test on GPU with
// parallelism on the track-level. It contains mainly two parts: 1) Explicit
// calling of the propagation to create measurements on tracks ( a 'simulated'
// track could contain 10~100 measurements) 2) Running the Kalmanfitter using
// the created measurements in 1) as one of the inputs In princinple, both 1)
// and 2) could on offloaded to GPU. Right now, only 2) is put into a kernel

static void show_usage(std::string name) {
  std::cerr << "Usage: <option(s)> VALUES"
            << "Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-t,--tracks \tSpecify the number of tracks\n"
            << "\t-e,--streams \tSpecify number of streams\n"
            << "\t-r,--threads \tSpecify the number of threads\n"
            << "\t-u,--multiple-devices \tIndicator for running on multiple GPUs (if available)\n"
            // << "\t-p,--pt \tSpecify the pt of particle\n"
            << "\t-o,--output \tIndicator for writing propagation results\n"
            << "\t-d,--device \tSpecify the device: 'gpu' or 'cpu'\n"
            << "\t-g,--grid-size \tSpecify GPU grid size: 'x*y'\n"
            << "\t-b,--block-size \tSpecify GPU block size: 'x*y*z'\n"
            << "\t-s,--shared-memory \tIndicator for using shared memory for "
               "one track or not\n"
            << "\t-m,--smoothing \tIndicator for running smoothing\n"
            << "\t-a,--machine \tThe name of the machine, e.g. V100\n"
            << std::endl;
}


int main(int argc, char *argv[]) {
  Size nTracks = 10000;
  Size nStreams = 1;
  Size nThreads = 250;
  Size nDevices = 1;
  // The number of navigation surfaces
  const Size nSurfaces = 10;
  bool output = false;
  bool useSharedMemory = false;
  bool smoothing = true;
  bool multiGpu = false;
  std::string device = "cpu";
  std::string machine;
  std::string bFieldFileName;
  // ActsScalar p = 1 * Acts::units::_GeV;
  dim3 grid(20000), block(8, 8);
  // This should always be included
  for (Size i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if ((arg == "-h") or (arg == "--help")) {
      show_usage(argv[0]);
      return 0;
    } else if (i + 1 < argc) {
      if ((arg == "-t") or (arg == "--tracks")) {
        nTracks = atoi(argv[++i]);
      } else if ((arg == "-e") or (arg == "--streams")) {
        nStreams = atoi(argv[++i]);
        if (multiGpu && nStreams > 1) {
          std::cerr << "--multiple-devices and --streams options are incompatible. Choose only one for now!" << std::endl;
          return 1;
        }
      } else if ((arg == "-r") or (arg == "--threads")) {
        nThreads = atoi(argv[++i]);
    //} else if ((arg == "-p") or (arg == "--pt")) {
     //  p = atof(argv[++i]) * Acts::units::_GeV;
      } else if ((arg == "-u") or (arg == "--multiple-devices")) {
        multiGpu = (atoi(argv[++i]) == 1);
        if (multiGpu) {
          if (nStreams > 1) { 
            std::cerr << "--multiple-devices and --streams options are incompatible. Choose only one for now!" << std::endl;
            return 1;
          } else  {
           int nDev;
           GPUERRCHK(hipGetDeviceCount(&nDev));
           nDevices = (Size)nDev;
           nStreams = nDevices;
          }
        }
      } else if ((arg == "-o") or (arg == "--output")) {
        output = (atoi(argv[++i]) == 1);
      } else if ((arg == "-d") or (arg == "--device")) {
        device = argv[++i];
      } else if ((arg == "-a") or (arg == "--machine")) {
        machine = argv[++i];
      } else if ((arg == "-g") or (arg == "--grid-size")) {
        grid = stringToDim3(argv[++i]);
      } else if ((arg == "-b") or (arg == "--block-size")) {
        block = stringToDim3(argv[++i]);
      } else if ((arg == "-s") or (arg == "--shared-memory")) {
        useSharedMemory = (atoi(argv[++i]) == 1);
      } else if ((arg == "-m") or (arg == "--smoothing")) {
        smoothing = (atoi(argv[++i]) == 1);
      } else {
        std::cerr << "Unknown argument." << std::endl;
        return 1;
      }
    }
  }

  if (grid.z != 1 or block.z != 1) {
    std::cout << "3D grid or block is not supported at the moment! Good luck!"
              << std::endl;
    return 1;
  }
  std::cout << grid.x << " " << grid.y << " " << block.x << " " << block.y
            << std::endl;

  std::cout << "Devices requested for KF: " << std::endl;

  hipDeviceProp_t prop;
  for (Size devId = 0; devId < nDevices; devId++) {
    GPUERRCHK(hipSetDevice(devId));
    GPUERRCHK(hipGetDeviceProperties(&prop, devId));
    printf("   Device : %s\n", prop.name);
    int driverVersion, rtVersion;
    GPUERRCHK(hipDriverGetVersion(&driverVersion));
    printf("   Cuda driver version: %i\n", driverVersion);
    GPUERRCHK(hipRuntimeGetVersion(&rtVersion));
    printf("   Cuda rt version: %i\n\n", rtVersion);
  }

  if (machine.empty()) {
    if (device == "gpu") {
      machine = prop.name;
      std::replace(machine.begin(), machine.end(), ' ', '_');
      if (multiGpu)
 	machine.append("x").append(std::to_string(nDevices));	
    } else {
      std::cout << "ERROR: The name of the CPU being tested must be provided, "
                   "like e.g. "
                   "Intel_i7-8559U."
                << std::endl;
      return 1;
    }
  }

  Size tracksPerBlock = block.x * block.y;

  // Use 8*8 block if using one block for one track
  if (useSharedMemory) {
    std::cout << "Shared memory used. Block size is set to 8*8!" << std::endl;
    block = dim3(8, 8);
    tracksPerBlock = 1;
  }

  // The navigation surfaces
  const Size navigationSurfaceBytes = sizeof(PlaneSurfaceType) * nSurfaces;
  // The track-specific objects
  const auto dataBytes = FitDataSizeCalculator::totalBytes(nSurfaces, nTracks);
  // The last stream could could less tracks
  const auto dataBytesPerStream =
      FitDataSizeCalculator::streamBytes(nSurfaces, nTracks, nStreams, 0);
  const auto dataBytesLastStream = FitDataSizeCalculator::streamBytes(
      nSurfaces, nTracks, nStreams, nStreams - 1);
  const Size tracksPerStream = dataBytesPerStream[7];
  const Size tracksLastStream = dataBytesLastStream[7];
  std::cout << "navigation surfaces Bytes = " << navigationSurfaceBytes
            << std::endl;
  for (const auto bytes : dataBytes) {
    std::cout << "dataBytes = " << bytes << std::endl;
  }

  std::cout << "tracksPerStream:tracksLastStream = " << tracksPerStream << " : "
            << tracksLastStream << std::endl;

  // @note shall we use this for the grid size?
  const Size blocksPerGrid =
      (tracksPerStream + tracksPerBlock - 1) / tracksPerBlock;
  if (grid.x * grid.y < blocksPerGrid) {
    std::cout << "WARNING: Grid size too small. It's set to the minimum size: "
              << blocksPerGrid << std::endl;
    grid = blocksPerGrid;
  }

  // The shared memory size
  //  using PropState = PropagatorType::State<PropOptionsType>;
  //  int sharedMemoryPerTrack =
  //      sizeof(Acts::PathLimitReached) + sizeof(PropState) + sizeof(bool) * 2
  //      + sizeof(Acts::PropagatorResult) + sizeof(Acts::ActsMatrixD<2,
  //      Acts::eBoundParametersSize>) * 2 + sizeof(Acts::ActsMatrixD<2, 2>) * 2
  //      + sizeof(Acts::BoundMatrix);
  //  std::cout << "shared memory is " << sharedMemoryPerTrack << std::endl;

  // Create a test context
  Acts::GeometryContext gctx(0);
  Acts::MagneticFieldContext mctx(0);

  // Create a random number service
  ActsExamples::RandomNumbers::Config config;
  auto randomNumbers = std::make_shared<ActsExamples::RandomNumbers>(config);
  auto rng = randomNumbers->spawnGenerator(0);

  // Create the geometry
  // Set translation vectors
  std::vector<Acts::Vector3D> translations;
  for (Size isur = 0; isur < nSurfaces; isur++) {
    Acts::Vector3D translation(isur * 30. + 20., 0., 0.);
    translations.emplace_back(translation);
  }
  // The silicon material
  Acts::MaterialSlab matProp(Test::makeSilicon(), 0.5 * Acts::units::_mm);
  Acts::HomogeneousSurfaceMaterial surfaceMaterial(matProp);
  // Create plane surfaces without boundaries
  PlaneSurfaceType *surfaces;
  // Unified memory allocation for geometry
  GPUERRCHK(hipHostMalloc(&surfaces, navigationSurfaceBytes));
  std::cout << "Allocating the memory for the surfaces" << std::endl;
  for (Size isur = 0; isur < nSurfaces; isur++) {
    surfaces[isur] = PlaneSurfaceType(translations[isur],
                                      Acts::Vector3D(1, 0, 0), surfaceMaterial);
    if (not surfaces[isur].surfaceMaterial().materialSlab()) {
      std::cerr << "No surface material" << std::endl;
    }
  }
  // Assign the geometry ID
  for (Size isur = 0; isur < nSurfaces; isur++) {
    auto geoID =
        Acts::GeometryID().setVolume(0u).setLayer(isur).setSensitive(isur);
    surfaces[isur].assignGeoID(geoID);
    // printf("surface value = %d, geoID = (%d, %d, %d)\n",
    //       surfaces[isur].geoID().value(), surfaces[isur].geoID().volume(),
    //       surfaces[isur].geoID().layer(),
    //       surfaces[isur].geoID().sensitive());
  }
  const Acts::Surface *surfacePtrs = surfaces;
  std::cout << "Creating " << nSurfaces << " boundless plane surfaces"
            << std::endl;

  // Pinned memory for data objects
  Acts::PixelSourceLink *sourcelinks;
  BoundState *boundStates;
  Acts::LineSurface *targetSurfaces;
  FitOptionsType *fitOptions;
  TSType *fitStates;
  Acts::BoundParameters<Acts::LineSurface> *fitPars;
  bool *fitStatus;
  GPUERRCHK(
      hipHostMalloc((void **)&sourcelinks, dataBytes[FitData::SourceLinks]));
  GPUERRCHK(
      hipHostMalloc((void **)&boundStates, dataBytes[FitData::StartState]));
  GPUERRCHK(hipHostMalloc((void **)&targetSurfaces,
                           dataBytes[FitData::TargetSurface]));
  GPUERRCHK(
      hipHostMalloc((void **)&fitOptions, dataBytes[FitData::FitOptions]));
  GPUERRCHK(hipHostMalloc((void **)&fitStates, dataBytes[FitData::FitStates]));
  GPUERRCHK(hipHostMalloc((void **)&fitPars, dataBytes[FitData::FitParams]));
  GPUERRCHK(hipHostMalloc((void **)&fitStatus, dataBytes[FitData::FitStatus]));

  // Prepare to run the particles generation
  ActsExamples::GaussianVertexGenerator vertexGen;
  vertexGen.stddev[Acts::eFreePos0] = 1.0 * Acts::units::_mm;
  vertexGen.stddev[Acts::eFreePos1] = 1.0 * Acts::units::_mm;
  vertexGen.stddev[Acts::eFreePos2] = 5.0 * Acts::units::_mm;
  vertexGen.stddev[Acts::eFreeTime] = 1.0 * Acts::units::_ns;
  ActsExamples::ParametricParticleGenerator::Config pgCfg;
  // @note We are generating 20% more particles to make sure we could get enough
  // valid particles
  size_t nGeneratedParticles = nTracks * 1.2;
  ActsExamples::Generator generator = ActsExamples::Generator{
      ActsExamples::FixedMultiplicityGenerator{nGeneratedParticles},
      std::move(vertexGen), ActsExamples::ParametricParticleGenerator(pgCfg)};
  // Run the generation to generate particles
  std::vector<ActsFatras::Particle> generatedParticles;
  runParticleGeneration(rng, generator, generatedParticles);

  // Prepare to run the simulation
  Stepper stepper;
  PropagatorType propagator(stepper);
  auto start_propagate = std::chrono::high_resolution_clock::now();
  std::cout << "start to run propagation" << std::endl;
  // Run the simulation to generate sim hits
  // @note We will pick up the valid particles
  std::vector<Simulator::result_type> simResult(nTracks);
  std::vector<ActsFatras::Particle> validParticles(nTracks);
  runSimulation(gctx, mctx, rng, propagator, generatedParticles, validParticles,
                simResult, surfacePtrs, nSurfaces);
  auto end_propagate = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_seconds =
      end_propagate - start_propagate;
  std::cout << "Time (ms) to run propagation tests: "
            << elapsed_seconds.count() * 1000 << std::endl;
  if (output) {
    std::cout << "writing propagation results" << std::endl;
    std::string simFileName =
        "sim_hits_for_" + std::to_string(nTracks) + "_particles.obj";
    writeSimHitsObj(simResult, simFileName);
  }

  // Build the target surfaces based on the truth particle position
  buildTargetSurfaces(validParticles, targetSurfaces);

  // The hit smearing resolution
  std::array<ActsScalar, 2> hitResolution = {30. * Acts::units::_mm,
                                             30. * Acts::units::_mm};
  // Run hit smearing to create source links
  // @note pass the concreate PlaneSurfaceType pointer here
  runHitSmearing(gctx, rng, simResult, hitResolution, sourcelinks, surfaces,
                 nSurfaces);

  // The particle smearing resolution
  ParticleSmearingParameters seedResolution;
  // Run truth seed smearing to create starting parameters with provided
  // reference surface
  auto startParsCollection = runParticleSmearing(
      rng, gctx, validParticles, seedResolution, targetSurfaces, nTracks);
  // Initialize the boundState
  for (Size it = 0; it < nTracks; it++) {
    boundStates[it].boundParams = startParsCollection[it].parameters();
    boundStates[it].boundCov = *startParsCollection[it].covariance();
  }

  // Prepare to perform fit to the created tracks
  KalmanFitterType kFitter(propagator);
  // Initialize the fitOptions and fit status
  for (Size it = 0; it < nTracks; it++) {
    fitOptions[it] = FitOptionsType(gctx, mctx, smoothing);
    fitStatus[it] = false;
  }

  float sec; // elapsed time in seconds

  // @note: prefetch the surface or not
  // hipMemPrefetchAsync(surfaces, navigationSurfaceBytes, devId, stream[0]);

  // Running directly on host or offloading to GPU
  bool useGPU = (device == "gpu");
  if (useGPU) {
    
    auto startFitTime = omp_get_wtime();

    // The same number of streams is available, but used either:
    // a. in parallel on the same device, OR
    // b. one per device, in parallel for all devices;
    hipStream_t stream[nStreams];
    Size max = std::max(nDevices,nStreams);

    #pragma omp parallel for 
    for (Size i = 0; i < max; ++i) {
         GPUERRCHK(hipSetDevice(multiGpu ? i : 0));
         GPUERRCHK(hipStreamCreate(&stream[i]));
    }
  
  #pragma omp parallel for num_threads(max) proc_bind(master)
  for (Size devId = 0; devId < nDevices; ++devId) {
        auto startDeviceTime = omp_get_wtime();

        // Set the corresponding device
        GPUERRCHK(hipSetDevice(devId));
      
        // Allocate memory on the device
        PlaneSurfaceType *d_surfaces;
        KalmanFitterType *d_kFitter;
        Acts::PixelSourceLink *d_sourcelinks;
        BoundState *d_boundStates;
        Acts::LineSurface *d_targetSurfaces;
        FitOptionsType *d_fitOptions;
        TSType *d_fitStates;
        Acts::BoundParameters<Acts::LineSurface> *d_fitPars;
        bool *d_fitStatus;
      
        GPUERRCHK(hipMalloc(&d_surfaces, navigationSurfaceBytes));
        GPUERRCHK(hipMalloc(&d_kFitter, sizeof(KalmanFitterType)));
        GPUERRCHK(hipMalloc(&d_sourcelinks, dataBytes[FitData::SourceLinks]));
        GPUERRCHK(hipMalloc(&d_boundStates, dataBytes[FitData::StartState]));
        GPUERRCHK(hipMalloc(&d_targetSurfaces, dataBytes[FitData::TargetSurface]));
        GPUERRCHK(hipMalloc(&d_fitOptions, dataBytes[FitData::FitOptions]));
        GPUERRCHK(hipMalloc(&d_fitStates, dataBytes[FitData::FitStates]));
        GPUERRCHK(hipMalloc(&d_fitPars, dataBytes[FitData::FitParams]));
        GPUERRCHK(hipMalloc(&d_fitStatus, dataBytes[FitData::FitStatus]));
      
        // Copy the KalmanFitter from host to device (shared between all tracks)
        GPUERRCHK(hipMemcpy(d_surfaces, surfaces, navigationSurfaceBytes,
                             hipMemcpyHostToDevice));
        GPUERRCHK(hipMemcpy(d_kFitter, &kFitter, sizeof(KalmanFitterType),
                             hipMemcpyHostToDevice));
       
        // If more devices are available, then there is only 1 stream per device;
        // If only 1 device is available, then there are nStreams streams used;
        Size streamStartIdx = multiGpu ? devId : 0;
        Size streamEndIdx = multiGpu ? (devId+1) : nStreams;

        for (Size i = streamStartIdx; i < streamEndIdx; ++i) {
	  GPUERRCHK(hipSetDevice(devId));
          Size offset = i * tracksPerStream;
          const auto streamTracks =
              (i < nStreams - 1) ? tracksPerStream : tracksLastStream;
          const auto streamDataBytes =
              (i < nStreams - 1) ? dataBytesPerStream : dataBytesLastStream;
      
          // Copy the sourcelinsk, starting parameters and fitted tracks from host
          // to device
          GPUERRCHK(hipMemcpyAsync(&d_sourcelinks[offset * nSurfaces],
                                    &sourcelinks[offset * nSurfaces],
                                    streamDataBytes[FitData::SourceLinks],
                                    hipMemcpyHostToDevice, stream[i]));
          GPUERRCHK(hipMemcpyAsync(&d_boundStates[offset], &boundStates[offset],
                                    streamDataBytes[FitData::StartState],
                                    hipMemcpyHostToDevice, stream[i]));
          GPUERRCHK(hipMemcpyAsync(&d_targetSurfaces[offset],
                                    &targetSurfaces[offset],
                                    streamDataBytes[FitData::TargetSurface],
                                    hipMemcpyHostToDevice, stream[i]));
          GPUERRCHK(hipMemcpyAsync(&d_fitOptions[offset], &fitOptions[offset],
                                    streamDataBytes[FitData::FitOptions],
                                    hipMemcpyHostToDevice, stream[i]));
          GPUERRCHK(hipMemcpyAsync(&d_fitStates[offset * nSurfaces],
                                    &fitStates[offset * nSurfaces],
                                    streamDataBytes[FitData::FitStates],
                                    hipMemcpyHostToDevice, stream[i]));
          GPUERRCHK(hipMemcpyAsync(&d_fitPars[offset], &fitPars[offset],
                                    streamDataBytes[FitData::FitParams],
                                    hipMemcpyHostToDevice, stream[i]));
          GPUERRCHK(hipMemcpyAsync(&d_fitStatus[offset], &fitStatus[offset],
                                    streamDataBytes[FitData::FitStatus],
                                    hipMemcpyHostToDevice, stream[i]));
      //    std::cout << "prepared to launch kernel\n" << std::endl;
          // Use shared memory for one track if requested
          if (useSharedMemory) {
            fitKernelBlockPerTrack<<<grid, block, 0, stream[i]>>>(
                d_kFitter, d_sourcelinks, d_boundStates, d_targetSurfaces,
                d_fitOptions, d_fitStates, d_fitPars, d_fitStatus, d_surfaces,
                nSurfaces, streamTracks, offset);
          } else {
            fitKernelThreadPerTrack<<<grid, block, 0, stream[i]>>>(
                d_kFitter, d_sourcelinks, d_boundStates, d_targetSurfaces,
                d_fitOptions, d_fitStates, d_fitPars, d_fitStatus, d_surfaces,
                nSurfaces, streamTracks, offset);
          }

          // copy the fitted states to host
          GPUERRCHK(hipMemcpyAsync(&fitStates[offset * nSurfaces],
                                    &d_fitStates[offset * nSurfaces],
                                    streamDataBytes[FitData::FitStates],
                                    hipMemcpyDeviceToHost, stream[i]));
          if (smoothing) {
            // copy the fitted params to host
            GPUERRCHK(hipMemcpyAsync(&fitPars[offset], &d_fitPars[offset],
                                      streamDataBytes[FitData::FitParams],
                                      hipMemcpyDeviceToHost, stream[i]));
          }
          // copy the fit status to host
          GPUERRCHK(hipMemcpyAsync(&fitStatus[offset], &d_fitStatus[offset],
                                    streamDataBytes[FitData::FitStatus],
                                    hipMemcpyDeviceToHost, stream[i]));
        }
        GPUERRCHK(hipPeekAtLastError());
        GPUERRCHK(hipDeviceSynchronize());
          
        // Free the memory on device
        GPUERRCHK(hipFree(d_sourcelinks));
        GPUERRCHK(hipFree(d_boundStates));
        GPUERRCHK(hipFree(d_targetSurfaces));
        GPUERRCHK(hipFree(d_fitOptions));
        GPUERRCHK(hipFree(d_fitStates));
        GPUERRCHK(hipFree(d_fitPars));
        GPUERRCHK(hipFree(d_fitStatus));
        GPUERRCHK(hipFree(d_kFitter));
        GPUERRCHK(hipFree(d_surfaces));
        
        for (Size i = streamStartIdx; i < streamEndIdx; i++) {
          GPUERRCHK(hipStreamDestroy(stream[i]));
        }
      
        auto stopDeviceTime = omp_get_wtime();
     
        printf("Thread %d: Time (ms) for KF memory transfer and execution on "
                 "device %d : %f\n",
                 omp_get_thread_num(), devId, (stopDeviceTime-startDeviceTime)*1000);            
    }

    auto endFitTime = omp_get_wtime();
    sec = endFitTime - startFitTime; 
    printf("Total Wall clock time (ms) for KF: %f\n", sec*1000);

    // Log the execution time in seconds (not including the managed memory
    // allocation time for the surfaces)
   Test::Logger::logTime(
        Test::Logger::buildFilename(
            "timing", machine, "nTracks", std::to_string(nTracks), "nStreams",
            std::to_string(nStreams), "gridSize", dim3ToString(grid),
            "blockSize", dim3ToString(block), "sharedMemory",
            std::to_string(static_cast<Size>(useSharedMemory))),
        sec*1000);

  } else {
    /// Test without GPU offloading
    int threads = 1;
    auto start_fit = std::chrono::high_resolution_clock::now();
#pragma omp parallel for num_threads(nThreads)
    for (Size it = 0; it < nTracks; it++) {
      // The fit result wrapper
      KalmanFitterResultType kfResult;
      kfResult.fittedStates = Acts::CudaKernelContainer<TSType>(
          &fitStates[it * nSurfaces], nSurfaces);
      // @note when it >=35, we got different startPars[i] between CPU and GPU
      // The input source links wrapper
      auto sourcelinkTrack = Acts::CudaKernelContainer<Acts::PixelSourceLink>(
          sourcelinks + it * nSurfaces, nSurfaces);
      fitOptions[it].referenceSurface = &targetSurfaces[it];
      // Run the fit. The fitStates will be changed here
      auto status = kFitter.fit(sourcelinkTrack, startParsCollection[it],
                                fitOptions[it], kfResult, surfaces, nSurfaces);
      if (not status) {
        std::cout << "fit failure for track " << it << std::endl;
      }
      // store the fit parameters and status
      fitStatus[it] = status;
      fitPars[it] = kfResult.fittedParameters;
      threads = omp_get_num_threads();
    }
    auto end_fit = std::chrono::high_resolution_clock::now();
    elapsed_seconds = end_fit - start_fit;
    std::cout << "Time (ms) to run KalmanFitter for " << nTracks << " : "
              << elapsed_seconds.count() * 1000 << std::endl;

    // Log execution time in csv file
    Test::Logger::logTime(
        Test::Logger::buildFilename("timing_semi", machine, "nTracks",
                                    std::to_string(nTracks), "OMP_NumThreads",
                                    std::to_string(threads)),
        elapsed_seconds.count() * 1000);
  }

  if (output) {
    std::cout << "writing KF results" << std::endl;
    std::string stateFileName;
    std::string csvFileName;
    std::string rootFileName;

    // The type of output parameters
    std::string state = smoothing ? "smoothed" : "filtered";
    stateFileName.append("fitted_");
    stateFileName.append(state);

    // The fitted parameters at the target surface
    csvFileName.append("fitted_");
    csvFileName.append("param");

    rootFileName.append("fitted_");
    rootFileName.append("param");

    // The type of machines
    std::string machine_prefix = useGPU ? "_" : "_semi_";
    stateFileName.append(machine_prefix);
    stateFileName.append(machine);

    csvFileName.append(machine_prefix);
    csvFileName.append(machine);

    rootFileName.append(machine_prefix);
    rootFileName.append(machine);

    // The number of tracks
    stateFileName.append("_nTracks_");
    csvFileName.append("_nTracks_");
    rootFileName.append("_nTracks_");

    // The type of the file written out
    stateFileName.append(std::to_string(nTracks)).append(".obj");
    csvFileName.append(std::to_string(nTracks)).append(".csv");
    rootFileName.append(std::to_string(nTracks)).append(".root");
    writeStatesObj(fitStates, fitStatus, nTracks, nSurfaces, stateFileName,
                   state);
    // The fitted parameters will be meaningful only after smoothing
    if (smoothing) {
      writeParamsCsv(fitPars, fitStatus, nTracks, csvFileName);
      writeParamsRoot(gctx, fitPars, fitStatus, validParticles, nTracks,
                      rootFileName, "params");
    }
  }

  std::cout << "------------------------  ending  -----------------------"
            << std::endl;

  // Free the managed/pinned memory
  GPUERRCHK(hipHostFree(surfaces));
  GPUERRCHK(hipHostFree(sourcelinks));
  GPUERRCHK(hipHostFree(boundStates));
  GPUERRCHK(hipHostFree(targetSurfaces));
  GPUERRCHK(hipHostFree(fitOptions));
  GPUERRCHK(hipHostFree(fitStates));
  GPUERRCHK(hipHostFree(fitPars));
  GPUERRCHK(hipHostFree(fitStatus));

  return 0;
}

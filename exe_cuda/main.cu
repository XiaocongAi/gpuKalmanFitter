#include "hip/hip_runtime.h"
#include "EventData/TrackParameters.hpp"
#include "Plugins/BFieldOptions.hpp"
#include "Plugins/BFieldUtils.hpp"
#include "Propagator/EigenStepper.hpp"
#include "Propagator/Propagator.hpp"

#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#define GPUERRCHK(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

static void show_usage(std::string name) {
  std::cerr << "Usage: <option(s)> VALUES"
            << "Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-t,--tracks \tSpecify the number of tracks\n"
            << "\t-p,--pt \tSpecify the pt of particle\n"
            << "\t-o,--output \tIndicator for writing propagation results\n"
            << "\t-d,--device \tSpecify the device: 'gpu' or 'cpu'\n"
            << "\t-b,--bf-map \tSpecify the path of *.txt for interpolated "
               "BField map\n"
            << std::endl;
}

using namespace Acts;

// Struct for B field
struct ConstantBField {
  ACTS_DEVICE_FUNC static Vector3D getField(const Vector3D & /*field*/) {
    return Vector3D(0., 0., 2.);
  }
};

constexpr unsigned int maxSteps = 1000;

//using Stepper = EigenStepper<ConstantBField>;
using Stepper = EigenStepper<InterpolatedBFieldMap3D>;
using PropagatorType = Propagator<Stepper>;
using PropResultType = PropagatorResult<maxSteps>;

// Device code
__global__ void propKernel(PropagatorType *propagator, TrackParameters *tpars,
                           PropagatorOptions *propOptions,
                           PropResultType *propResult, Vector3D *gridValPtr,
                           int N) {
  // Awkwardly make the grid values pointer to point to memeory on device
  // explicitly
  propagator->refStepper().refField().refMapper().refGrid().refValues() =
      gridValPtr;

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    propagator->propagate(tpars[i], *propOptions, propResult[i]);
    // printf("propResult: position = (%f, %f, %f)",
    // propResult[i].position.col(1).x(), propResult[i].position.col(1).y(),
    // propResult[i].position.col(1).z());
  }
}

int main(int argc, char *argv[]) {
  if (argc < 5) {
    show_usage(argv[0]);
    return 1;
  }
  unsigned int nTracks;
  bool output = false;
  std::string device;
  std::string bFieldFileName;
  double p;
  for (int i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if ((arg == "-h") or (arg == "--help")) {
      show_usage(argv[0]);
      return 0;
    } else if (i + 1 < argc) {
      if ((arg == "-t") or (arg == "--tracks")) {
        nTracks = atoi(argv[++i]);
      } else if ((arg == "-p") or (arg == "--pt")) {
        p = atof(argv[++i]);
      } else if ((arg == "-o") or (arg == "--output")) {
        output = (atoi(argv[++i]) == 1);
      } else if ((arg == "-d") or (arg == "--device")) {
        device = argv[++i];
      } else if ((arg == "-b") or (arg == "--bf-map")) {
        bFieldFileName = argv[++i];
      } else {
        std::cerr << "Unknown argument." << std::endl;
        return 1;
      }
    }
  }

  std::cout << "----- Propgation test of " << nTracks << " tracks on " << device
            << ". Writing results to obj file? " << output << " ----- "
            << std::endl;

  InterpolatedBFieldMap3D bField = Options::readBField(bFieldFileName);
  std::cout
      << "Reading BField and creating a 3D InterpolatedBFieldMap instance done"
      << std::endl;

  // Construct a stepper with the bField
  Stepper stepper(bField);
  // Construct a propagator
  PropagatorType propagator(stepper);
  // Construct the propagation options object
  PropagatorOptions propOptions;
  propOptions.maxSteps = 1000;
  propOptions.maxStepSize = 1000;

  // Construct random starting track parameters
  std::default_random_engine generator(42);
  std::normal_distribution<double> gauss(0., 1.);
  std::uniform_real_distribution<double> unif(-1.0 * M_PI, M_PI);
  std::vector<TrackParameters> pars;
  pars.reserve(nTracks);
  for (int i = 0; i < nTracks; i++) {
    Vector3D rPos(0.1 * gauss(generator), 0.1 * gauss(generator),
                  0); // Units: mm
    double phi =  unif(generator);
    double theta = M_PI/2 + gauss(generator)*0.01;
    Vector3D rMom(p*sin(theta)*cos(phi), p*sin(theta)*sin(phi), p*cos(theta)); // Units: GeV
    double q = 1;
    TrackParameters rStart(rPos, rMom, q);
    pars[i] = rStart;
  }

  // Propagation result
  std::vector<PropResultType> ress;
  ress.reserve(nTracks);

  auto start = std::chrono::high_resolution_clock::now();

  // Running directly on host or offloading to GPU
  bool useGPU = (device == "gpu" ? true : false);
  if (useGPU) {
    // We have to use a really nasty deep reference when dynamic allocation is
    // used for the grid values which cannot be automatically done on GPU?
    auto &grid = propagator.refStepper().refField().refMapper().refGrid();
    // Get the grid size and values (pointer)
    size_t gridSize = grid.size();
    using GridType = std::remove_reference<decltype(grid)>::type;
    using GridValueType = typename GridType::value_type;
    GridValueType* gridValPtr = grid.refValues();

    // Allocate memory on device
    PropagatorType *d_propagator;
    PropagatorOptions *d_opt;
    TrackParameters *d_pars;
    PropResultType *d_ress;
    GridValueType *d_gridValPtr;

    GPUERRCHK(hipMalloc(&d_propagator, sizeof(PropagatorType)));
    GPUERRCHK(hipMalloc(&d_opt, sizeof(PropagatorOptions)));
    GPUERRCHK(hipMalloc(&d_pars, nTracks * sizeof(TrackParameters)));
    GPUERRCHK(hipMalloc(&d_ress, nTracks * sizeof(PropResultType)));
    GPUERRCHK(hipMalloc(&d_gridValPtr, gridSize * sizeof(GridValueType)));

    // Copy from host to device
    GPUERRCHK(hipMemcpy(d_propagator, &propagator, sizeof(propagator),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_opt, &propOptions, sizeof(PropagatorOptions),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_pars, pars.data(), nTracks * sizeof(TrackParameters),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_ress, ress.data(), nTracks * sizeof(PropResultType),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_gridValPtr, gridValPtr, gridSize * sizeof(GridValueType),
                         hipMemcpyHostToDevice));

    // Run on device
    int threadsPerBlock = 256;
    int blocksPerGrid = (nTracks + threadsPerBlock - 1) / threadsPerBlock;
    propKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_propagator, d_pars, d_opt, d_ress, d_gridValPtr, nTracks);

    GPUERRCHK(hipPeekAtLastError());
    GPUERRCHK(hipDeviceSynchronize());

    // Copy result from device to host
    GPUERRCHK(hipMemcpy(ress.data(), d_ress, nTracks * sizeof(PropResultType),
                         hipMemcpyDeviceToHost));

    // Free the memory on device
    GPUERRCHK(hipFree(d_propagator));
    GPUERRCHK(hipFree(d_opt));
    GPUERRCHK(hipFree(d_pars));
    GPUERRCHK(hipFree(d_ress));
    GPUERRCHK(hipFree(d_gridValPtr));
  } else {
    // Run on host
    #pragma omp parallel for
    for (int it = 0; it < nTracks; it++) {
      propagator.propagate(pars[it], propOptions, ress[it]);
    }
  }
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  std::cout << "Time (sec) to run propagation tests: "
            << elapsed_seconds.count() << std::endl;

  if (output) {
    // Write result to obj file
    std::cout << "Writing yielded " << nTracks << " tracks to obj files..."
              << std::endl;

    for (int it = 0; it < nTracks; it++) {
      PropResultType res = ress[it];
      std::ofstream obj_track;
      std::string fileName =
          device + "_output/Track-" + std::to_string(it) + ".obj";
      obj_track.open(fileName.c_str());

      for (int iv = 0; iv < res.steps(); iv++) {
        obj_track << "v " << res.position.col(iv).x() << " "
                  << res.position.col(iv).y() << " " << res.position.col(iv).z()
                  << std::endl;
      }
      for (unsigned int iv = 2; iv <= res.steps(); ++iv) {
        obj_track << "l " << iv - 1 << " " << iv << std::endl;
      }

      obj_track.close();
    }
  }

  std::cout << "------------------------  ending  -----------------------"
            << std::endl;

  return 0;
}

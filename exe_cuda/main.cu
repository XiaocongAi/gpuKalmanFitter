#include "hip/hip_runtime.h"
#include "Geometry/GeometryContext.hpp"
#include "MagneticField/MagneticFieldContext.hpp"
#include "Plugins/BFieldOptions.hpp"
#include "Plugins/BFieldUtils.hpp"
#include "Propagator/EigenStepper.hpp"
#include "EventData/TrackParameters.hpp"
#include "Propagator/Propagator.hpp"
#include "Utilities/ParameterDefinitions.hpp"
#include "Utilities/Units.hpp"

#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#define GPUERRCHK(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

static void show_usage(std::string name) {
  std::cerr << "Usage: <option(s)> VALUES"
            << "Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-t,--tracks \tSpecify the number of tracks\n"
            << "\t-p,--pt \tSpecify the pt of particle\n"
            << "\t-o,--output \tIndicator for writing propagation results\n"
            << "\t-d,--device \tSpecify the device: 'gpu' or 'cpu'\n"
            << std::endl;
}

using namespace Acts;

// Struct for B field
struct ConstantBField {
  ACTS_DEVICE_FUNC static Vector3D getField(const Vector3D & /*field*/) {
    return Vector3D(0., 0., 2.*Acts::units::_T);
  }
};

// Test actor
struct VoidActor {
  struct this_result {
    bool status = false;
  };
  using result_type = this_result;

  template <typename propagator_state_t, typename stepper_t>
  __host__ __device__ void operator()(propagator_state_t &state, const stepper_t &stepper,
                  result_type &result) const {
    return;
  }
};

// Test aborter
struct VoidAborter {
  template <typename propagator_state_t, typename stepper_t, typename result_t>
  __host__ __device__ bool operator()(propagator_state_t &state, const stepper_t &stepper,
                  result_t &result) const {
    return false;
  }
};

using Stepper = EigenStepper<ConstantBField>;
//using Stepper = EigenStepper<InterpolatedBFieldMap3D>;
using PropagatorType = Propagator<Stepper>;
using PropResultType =
    PropagatorResult<typename VoidActor::result_type>;
using PropOptionsType = PropagatorOptions<VoidActor, VoidAborter>;

// Device code
__global__ void propKernel(PropagatorType *propagator,
                           CurvilinearParameters *tpars,
                           //PropOptionsType *propOptions,
                           PropOptionsType propOptions,
                           PropResultType *propResult, 
                           int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    propResult[i] = propagator->propagate(tpars[i], propOptions);
  }
}

int main(int argc, char *argv[]) {
  if (argc < 5) {
    show_usage(argv[0]);
    return 1;
  }
  unsigned int nTracks;
  bool output = false;
  std::string device;
  std::string bFieldFileName;
  double p;
  for (int i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if ((arg == "-h") or (arg == "--help")) {
      show_usage(argv[0]);
      return 0;
    } else if (i + 1 < argc) {
      if ((arg == "-t") or (arg == "--tracks")) {
        nTracks = atoi(argv[++i]);
      } else if ((arg == "-p") or (arg == "--pt")) {
        p = atof(argv[++i])*Acts::units::_GeV;
      } else if ((arg == "-o") or (arg == "--output")) {
        output = (atoi(argv[++i]) == 1);
      } else if ((arg == "-d") or (arg == "--device")) {
        device = argv[++i];
      } else {
        std::cerr << "Unknown argument." << std::endl;
        return 1;
      }
    }
  }

   // Create the geometry
  size_t nSurfaces = 15; 
  // Set translation vectors
  std::vector<Acts::Vector3D> translations;
  for(unsigned int isur = 0; isur< nSurfaces; isur++){
    translations.push_back({(isur * 30. + 19)*Acts::units::_mm, 0., 0.});
  }

  Acts::PlaneSurface* surfaces;
  GPUERRCHK(hipMallocManaged(&surfaces, sizeof(Acts::PlaneSurface)*nSurfaces));
  for(unsigned int isur = 0; isur< nSurfaces; isur++){
    surfaces[isur] = Acts::PlaneSurface(translations[isur], Acts::Vector3D(1,0,0));
  }

  const Acts::Surface* surfacePtrs[nSurfaces];
  for(unsigned int isur = 0; isur< nSurfaces; isur++){
    surfacePtrs[isur] = &surfaces[isur];
  }

  std::cout<<"Creating "<<nSurfaces<<" boundless plane surfaces"<<std::endl;

  std::cout << "----- Propgation test of " << nTracks << " tracks on " << device
            << ". Writing results to obj file? " << output << " ----- "
            << std::endl;

  // Create a test context
  GeometryContext gctx;
  MagneticFieldContext mctx;

  //InterpolatedBFieldMap3D bField = Options::readBField(bFieldFileName);
  //std::cout
  //    << "Reading BField and creating a 3D InterpolatedBFieldMap instance done"
  //    << std::endl;

  // Construct a stepper with the bField
  Stepper stepper;
  PropagatorType propagator(stepper);
  PropOptionsType propOptions(gctx, mctx);
  propOptions.maxSteps = 10;
  propOptions.initializer.surfaceSequence = surfacePtrs;
  propOptions.initializer.surfaceSequenceSize = nSurfaces;

  // Construct random starting track parameters
  std::default_random_engine generator(42);
  std::normal_distribution<double> gauss(0., 1.);
  std::vector<CurvilinearParameters> startPars;
  startPars.reserve(nTracks);
  for (int i = 0; i < nTracks; i++) {
    BoundSymMatrix cov = BoundSymMatrix::Zero();
    cov << 0.01, 0., 0., 0., 0., 0., 0., 0.01, 0., 0., 0., 0., 0., 0., 0.0001,
        0., 0., 0., 0., 0., 0., 0.0001, 0., 0., 0., 0., 0., 0., 0.0001, 0., 0.,
        0., 0., 0., 0., 1.;

    double q = 1;
    double time = 0;
     double phi = gauss(generator)*0.01;
    double theta = M_PI/2 + gauss(generator)*0.01;
    Vector3D pos(-0, 0.1 * gauss(generator), 0.1 * gauss(generator)); // Units: mm
    Vector3D mom(p*sin(theta)*cos(phi), p*sin(theta)*sin(phi), p*cos(theta)); // Units: GeV 

    startPars.emplace_back(cov, pos, mom, q, time);
  }

  // Propagation result
  std::vector<PropResultType> ress;
  ress.reserve(nTracks);

  auto start = std::chrono::high_resolution_clock::now();

  // Running directly on host or offloading to GPU
  bool useGPU = (device == "gpu" ? true : false);
  if (useGPU) {
    // Allocate memory on device
    PropagatorType *d_propagator;
    //PropOptionsType *d_opt;
    CurvilinearParameters *d_pars;
    PropResultType *d_ress;

    GPUERRCHK(hipMalloc(&d_propagator, sizeof(PropagatorType)));
//    GPUERRCHK(hipMalloc(&d_opt, sizeof(PropOptionsType)));
    GPUERRCHK(hipMalloc(&d_pars, nTracks * sizeof(CurvilinearParameters)));
    GPUERRCHK(hipMalloc(&d_ress, nTracks * sizeof(PropResultType)));

    // Copy from host to device
    GPUERRCHK(hipMemcpy(d_propagator, &propagator, sizeof(propagator),
                         hipMemcpyHostToDevice));
//    GPUERRCHK(hipMemcpy(d_opt, &propOptions, sizeof(PropOptionsType),
//                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_pars, startPars.data(),
                         nTracks * sizeof(CurvilinearParameters),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_ress, ress.data(), nTracks * sizeof(PropResultType),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_ress, ress.data(), nTracks * sizeof(PropResultType),
                         hipMemcpyHostToDevice));

    // Run on device
    int threadsPerBlock = 256;
    int blocksPerGrid = (nTracks + threadsPerBlock - 1) / threadsPerBlock;
    propKernel<<<blocksPerGrid, threadsPerBlock>>>(
        //d_propagator, d_pars, d_opt, d_ress, nTracks);
        d_propagator, d_pars, propOptions, d_ress, nTracks);

    GPUERRCHK(hipPeekAtLastError());
    GPUERRCHK(hipDeviceSynchronize());

    // Copy result from device to host
    GPUERRCHK(hipMemcpy(ress.data(), d_ress, nTracks * sizeof(PropResultType),
                         hipMemcpyDeviceToHost));

    // Free the memory on device
    GPUERRCHK(hipFree(d_propagator));
//    GPUERRCHK(hipFree(d_opt));
    GPUERRCHK(hipFree(d_pars));
    GPUERRCHK(hipFree(d_ress));
    GPUERRCHK(hipFree(surfaces));
  } else {
// Run on host
#pragma omp parallel for
    for (int it = 0; it < nTracks; it++) {
      ress[it] = propagator.propagate(startPars[it], propOptions);
    }
  }
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  std::cout << "Time (sec) to run propagation tests: "
            << elapsed_seconds.count() << std::endl;

  if (output) {
    // Write result to obj file
    std::cout << "Writing yielded " << nTracks << " tracks to obj files..."
              << std::endl;

    for (int it = 0; it < nTracks; it++) {
      PropResultType res = ress[it];
      std::ofstream obj_track;
      std::string fileName =
          device + "_output/Track-" + std::to_string(it) + ".obj";
      obj_track.open(fileName.c_str());

      obj_track.close();
    }
  }

  std::cout << "------------------------  ending  -----------------------"
            << std::endl;

  return 0;
}

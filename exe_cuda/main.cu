#include "hip/hip_runtime.h"
#include "EventData/PixelSourceLink.hpp"
#include "EventData/TrackParameters.hpp"
#include "Fitter/GainMatrixUpdater.hpp"
#include "Fitter/KalmanFitter.hpp"
#include "Geometry/GeometryContext.hpp"
#include "MagneticField/MagneticFieldContext.hpp"
#include "Plugins/BFieldOptions.hpp"
#include "Plugins/BFieldUtils.hpp"
#include "Propagator/EigenStepper.hpp"
#include "Propagator/Propagator.hpp"
#include "Utilities/ParameterDefinitions.hpp"
#include "Utilities/Units.hpp"
#include "Test/TestHelper.hpp"

#include "Utilities/Profiling.hpp"

#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

// This executable is used to run the KalmanFitter fit test on GPU with parallelism on the track-level.
// It contains mainly two parts:
// 1) Explicit calling of the propagation to create measurements on tracks ( a 'simulated' track could contain 10~100 measurements)
// 2) Running the Kalmanfitter using the created measurements in 1) as one of the inputs
// In princinple, both 1) and 2) could on offloaded to GPU. Right now, only 2) is put into a kernel

#define GPUERRCHK(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

static void show_usage(std::string name) {
  std::cerr << "Usage: <option(s)> VALUES"
            << "Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-t,--tracks \tSpecify the number of tracks\n"
            << "\t-p,--pt \tSpecify the pt of particle\n"
            << "\t-o,--output \tIndicator for writing propagation results\n"
            << "\t-d,--device \tSpecify the device: 'gpu' or 'cpu'\n"
            << std::endl;
}

using namespace Acts;

using Stepper = EigenStepper<ConstantBField>;
// using Stepper = EigenStepper<InterpolatedBFieldMap3D>;
using PropagatorType = Propagator<Stepper>;
using PropResultType = PropagatorResult;
using PropOptionsType = PropagatorOptions<MeasurementCreator, VoidAborter>;

using KalmanFitterType = KalmanFitter<PropagatorType, GainMatrixUpdater>;
using KalmanFitterResultType =
    KalmanFitterResult<PixelSourceLink, BoundParameters>;
using TSType = typename KalmanFitterResultType::TrackStateType;

// Device code
__global__ void fitKernel(KalmanFitterType *kFitter,
                           PixelSourceLink *sourcelinks,
                           CurvilinearParameters *tpars,
                           KalmanFitterOptions<VoidOutlierFinder> kfOptions,
                           TSType *fittedTracks, const Surface *surfacePtrs,
                           int nSurfaces, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    // Use the CudaKernelContainer for the source links and fitted tracks
    KalmanFitterResultType kfResult;
    kfResult.fittedStates =
        CudaKernelContainer<TSType>(fittedTracks + i * nSurfaces, nSurfaces);
    kFitter->fit(CudaKernelContainer<PixelSourceLink>(
                     sourcelinks + i * nSurfaces, nSurfaces),
                 tpars[i], kfOptions, kfResult, surfacePtrs, nSurfaces);
  }
}

int main(int argc, char *argv[]) {
  if (argc < 5) {
    show_usage(argv[0]);
    return 1;
  }
  unsigned int nTracks = 1000;
  bool output = false;
  std::string device = "cpu";
  std::string bFieldFileName;
  double p = 1 * Acts::units::_GeV;
  for (int i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if ((arg == "-h") or (arg == "--help")) {
      show_usage(argv[0]);
      return 0;
    } else if (i + 1 < argc) {
      if ((arg == "-t") or (arg == "--tracks")) {
        nTracks = atoi(argv[++i]);
      } else if ((arg == "-p") or (arg == "--pt")) {
        p = atof(argv[++i]) * Acts::units::_GeV;
      } else if ((arg == "-o") or (arg == "--output")) {
        output = (atoi(argv[++i]) == 1);
      } else if ((arg == "-d") or (arg == "--device")) {
        device = argv[++i];
      } else {
        std::cerr << "Unknown argument." << std::endl;
        return 1;
      }
    }
  }

  // Create a test context
  GeometryContext gctx(0);
  MagneticFieldContext mctx(0);

  // Create the geometry
  size_t nSurfaces = 10;

  // Set translation vectors
  std::vector<Acts::Vector3D> translations;
  for (unsigned int isur = 0; isur < nSurfaces; isur++) {
    translations.push_back({(isur * 30. + 19) * Acts::units::_mm, 0., 0.});
  }

  // Acts::PlaneSurface surfaces[nSurfaces];
  Acts::PlaneSurface *surfaces;
  // Unifited memory allocation for geometry
  GPUERRCHK(
      hipMallocManaged(&surfaces, sizeof(Acts::PlaneSurface) * nSurfaces));
  std::cout << "Allocating the memory for the surfaces" << std::endl;
  for (unsigned int isur = 0; isur < nSurfaces; isur++) {
    surfaces[isur] =
        Acts::PlaneSurface(translations[isur], Acts::Vector3D(1, 0, 0));
  }
  std::cout << "Creating " << nSurfaces << " boundless plane surfaces"
            << std::endl;

  // Test the pointers to surfaces
  for (unsigned int isur = 0; isur < nSurfaces; isur++) {
    auto surface = surfaces[isur];
    std::cout << "surface " << isur << " has center at: \n"
              << surface.center(gctx) << std::endl;
  }

  std::cout << "----- Starting Kalman fitter test of " << nTracks
            << " tracks on " << device << std::endl;

  Acts::Surface* surfacePtrs = surfaces;

  // InterpolatedBFieldMap3D bField = Options::readBField(bFieldFileName);

  // Construct a stepper with the bField
  Stepper stepper;
  PropagatorType propagator(stepper);
  PropOptionsType propOptions(gctx, mctx);
  propOptions.maxSteps = 100;
  propOptions.initializer.surfaceSequence = surfacePtrs;
  propOptions.initializer.surfaceSequenceSize = nSurfaces;

  // Construct random starting track parameters
  std::vector<CurvilinearParameters> startPars;
  double resLoc1 = 0.1 * Acts::units::_mm;
  double resLoc2 = 0.1 * Acts::units::_mm;
  double resPhi = 0.01;
  double resTheta = 0.01;
  
  const BoundSymMatrix cov = [=] () {
    BoundSymMatrix cov = BoundSymMatrix::Zero();
    cov << resLoc1 * resLoc1, 0.,                0.,              0.,                  0.,     0.,
           0.,                resLoc2 * resLoc2, 0.,              0.,                  0.,     0.,
           0.,                0.,                resPhi * resPhi, 0.,                  0.,     0.,
           0.,                0.,                0.,              resTheta * resTheta, 0.,     0.,
           0.,                0.,                0.,              0.,                  0.0001, 0.,
           0.,                0.,                0.,              0.,                  0.,     1.;
    return cov;
  }();
	
  for (int i = 0; i < nTracks; i++) {
    
    double q = 1;
    double time = 0;
    double phi = gauss(generator) * resPhi;
    double theta = M_PI / 2 + gauss(generator) * resTheta;
    Vector3D pos(0, resLoc1 * gauss(generator),
                 resLoc2 * gauss(generator)); // Units: mm
    Vector3D mom(p * sin(theta) * cos(phi), p * sin(theta) * sin(phi),
                 p * cos(theta)); // Units: GeV

    startPars.emplace_back(cov, pos, mom, q, time);
  }
  std::cout << "Finish creating starting parameters" << std::endl;

  // Propagation result
  std::vector<MeasurementCreator::result_type> ress(nTracks);

  std::cout << "Start to run propagation to create measurements" << std::endl;
  auto start_propagate = std::chrono::high_resolution_clock::now();

  // Run propagation to create the measurements
  #pragma omp parallel for
  for (int it = 0; it < nTracks; it++) {
    propagator.propagate(startPars[it], propOptions, ress[it]);
  }

  auto end_propagate = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_seconds =
      end_propagate - start_propagate;
  std::cout << "Time (sec) to run propagation tests: "
            << elapsed_seconds.count() << std::endl;

  // Initialize the vertex counter
  unsigned int vCounter = 0;
  if (output) {
    std::cout << "writing propagation results" << std::endl;
    // Write all of the created tracks to one obj file
    std::ofstream obj_track;
    std::string fileName = "Tracks-propagation.obj";
    obj_track.open(fileName.c_str());

    for (int it = 0; it < nTracks; it++) {
      auto tracks = ress[it].sourcelinks;
      ++vCounter;
      for (const auto &sl : tracks) {
        const auto &pos = sl.globalPosition(gctx);
        obj_track << "v " << pos.x() << " " << pos.y() << " " << pos.z()
                  << "\n";
      }
      // Write out the line - only if we have at least two points created
      size_t vBreak = vCounter + tracks.size() - 1;
      for (; vCounter < vBreak; ++vCounter)
        obj_track << "l " << vCounter << " " << vCounter + 1 << '\n';
    }
    obj_track.close();
  }

  // Prepare to perform fit to the created tracks
  // Restore the source links
  std::vector<PixelSourceLink> sourcelinks(nSurfaces * nTracks);
  for (int it = 0; it < nTracks; it++) {
    const auto &sls = ress[it].sourcelinks;
    for (int is = 0; is < nSurfaces; is++) {
      sourcelinks[it * nSurfaces + is] = sls[is];
    }
  }

  // Create an KFitter
  PropagatorType rPropagator(stepper);
  KalmanFitterType kFitter(rPropagator);

  // The KF options
  KalmanFitterOptions<VoidOutlierFinder> kfOptions(gctx, mctx);

  // Allocate memory for KF fitted tracks
  TSType* fittedTracks;
  GPUERRCHK(hipMallocManaged(&fittedTracks, sizeof(TSType) * nSurfaces * nTracks));

  auto start_fit = std::chrono::high_resolution_clock::now();

  // Running directly on host or offloading to GPU
  bool useGPU = (device == "gpu" ? true : false);
  if (useGPU) {
    // Allocate memory on device
    PixelSourceLink *d_sourcelinks;
    CurvilinearParameters *d_pars;
    KalmanFitterType *d_kFitter;
    GPUERRCHK(hipMalloc(&d_sourcelinks,
                         sizeof(PixelSourceLink) * nSurfaces * nTracks));
    GPUERRCHK(hipMalloc(&d_pars, sizeof(CurvilinearParameters) * nTracks));
    GPUERRCHK(hipMalloc(&d_kFitter, sizeof(KalmanFitterType)));

    // Copy from host to device
    GPUERRCHK(hipMemcpy(d_sourcelinks, sourcelinks.data(),
                         sizeof(PixelSourceLink) * nSurfaces * nTracks,
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_pars, startPars.data(),
                         nTracks * sizeof(CurvilinearParameters),
                         hipMemcpyHostToDevice));
    GPUERRCHK(hipMemcpy(d_kFitter, &kFitter, sizeof(KalmanFitterType),
                         hipMemcpyHostToDevice));

    // Run on device
    int threadsPerBlock = 256;
    int blocksPerGrid = (nTracks + threadsPerBlock - 1) / threadsPerBlock;
    // Pass kfOptions by value

    for (int _ : {1, 2, 3, 4, 5}) {
    
    fitKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_kFitter, d_sourcelinks, d_pars, kfOptions, fittedTracks,
        surfacePtrs, nSurfaces, nTracks);
		
    }
    GPUERRCHK(hipPeekAtLastError());
    GPUERRCHK(hipDeviceSynchronize());
    
    // Free the memory on device
    GPUERRCHK(hipFree(d_sourcelinks));
    GPUERRCHK(hipFree(d_pars));
    GPUERRCHK(hipFree(d_kFitter));
    // GPUERRCHK(hipFree(surfacePtrs));
    GPUERRCHK(hipFree(surfaces));
    
  } else {
//// Run on host
#pragma omp parallel for
    for (int it = 0; it < nTracks; it++) {
      //     BoundSymMatrix cov = BoundSymMatrix::Zero();
      //     cov << resLoc1 * resLoc1, 0., 0., 0., 0., 0., 0., resLoc2 *
      //     resLoc2, 0.,
      //         0., 0., 0., 0., 0., resPhi * resPhi, 0., 0., 0., 0., 0., 0.,
      //         resTheta * resTheta, 0., 0., 0., 0., 0., 0., 0.0001, 0., 0.,
      //         0., 0., 0., 0., 1.;

      //     double q = 1;
      //     double time = 0;
      //     Vector3D pos(0, 0, 0); // Units: mm
      //     Vector3D mom(p, 0, 0); // Units: GeV

      //     CurvilinearParameters rStart(cov, pos, mom, q, time);

      // Dynamically allocating memory for the fitted states here
      KalmanFitterResultType kfResult;
      kfResult.fittedStates = CudaKernelContainer<TSType>(
          &fittedTracks[it * nSurfaces], nSurfaces);

      auto sourcelinkTrack = CudaKernelContainer<PixelSourceLink>(
          ress[it].sourcelinks.data(), ress[it].sourcelinks.size());

      // The fittedTracks will be changed here
      // Note that we are using exacty the truth starting parameters here (which
      // should be added smearing)
      auto fitStatus = kFitter.fit(sourcelinkTrack, startPars[it], kfOptions,
                                   kfResult, surfacePtrs, nSurfaces);
      if (not fitStatus) {
        std::cout << "fit failure for track " << it << std::endl;
      }
    }
  }

  auto end_fit = std::chrono::high_resolution_clock::now();
  elapsed_seconds = end_fit - start_fit;
  std::cout << "Time (sec) to run KalmanFitter for " << nTracks << " : "
            << elapsed_seconds.count() << std::endl;

  if (output) {
    std::cout << "writing KF results" << std::endl;
    // Write all of the created tracks to one obj file
    std::ofstream obj_ftrack;
    std::string fileName_ = "Tracks-fitted.obj";
    obj_ftrack.open(fileName_.c_str());

    // Initialize the vertex counter
    vCounter = 0;
    for (int it = 0; it < nTracks; it++) {
      ++vCounter;
      for (int is = 0; is < nSurfaces; is++) {
        const auto &pos =
            fittedTracks[it * nSurfaces + is].parameter.filtered.position();
        obj_ftrack << "v " << pos.x() << " " << pos.y() << " " << pos.z()
                   << "\n";
      }
      // Write out the line - only if we have at least two points created
      size_t vBreak = vCounter + nSurfaces - 1;
      for (; vCounter < vBreak; ++vCounter)
        obj_ftrack << "l " << vCounter << " " << vCounter + 1 << '\n';
    }
    obj_ftrack.close();
  }

  std::cout << "------------------------  ending  -----------------------"
            << std::endl;

  GPUERRCHK(hipFree(fittedTracks));

  return 0;
}
